#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <stdio.h>

__global__ void helloKernel(char* string) {
    int i = threadIdx.x;
	printf("%c", string[i]);
}

int main() {
	const char* msg = "Hello, CUDA world!";
	const int msglen = strlen(msg);
	char* dev_msg = NULL;

    hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	checkCudaErrors(cudaStatus);

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_msg, msglen);
	checkCudaErrors(cudaStatus);

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_msg, msg, msglen, hipMemcpyHostToDevice);
	checkCudaErrors(cudaStatus);

	// Launch a kernel on the GPU with one thread for each element.
	helloKernel <<<1, msglen>>>(dev_msg);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	checkCudaErrors(cudaStatus);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	checkCudaErrors(cudaStatus);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
	checkCudaErrors(cudaStatus);

	hipFree(dev_msg);

    return 0;
}
