#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <stdio.h>

hipEvent_t start, stop;
void startTimer() {
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
}

/** Return elapsed time (in ms) since startTime() was called */
float stopTimer() {
	float time;
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	return time;
}

const int ROWS = 32;
const int COLUMNS = 32;
int hostMatrix[ROWS][COLUMNS];

__global__ void transpose(int* in, int* out) {
	int i = (blockDim.x * threadIdx.y) + threadIdx.x;
	int o = (blockDim.y * threadIdx.x) + threadIdx.y;

	int tmp = in[i];
	out[o] = tmp;
}

__global__ void transposeShared(int* in, int* out) {
	int i = (blockDim.x * threadIdx.y) + threadIdx.x;
	int o = (blockDim.y * threadIdx.x) + threadIdx.y;

	__shared__ int s[ROWS][COLUMNS+1];

	int tmp = in[i];
	
	s[threadIdx.y][threadIdx.x] = tmp;

	__syncthreads();

	out[i] = s[threadIdx.x][threadIdx.y];
}

void printMatrix(int* mat, int rows, int cols) {
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			printf("%d ", mat[(i*cols) + j]);
		}
		printf("\r\n");
	}
	printf("\r\n");
}

int main() {

	for (int i = 0; i < ROWS; i++) {
		for (int j = 0; j < COLUMNS; j++) {
			hostMatrix[i][j] = (i * COLUMNS) + j;
		}
	}

	int* dev_in = NULL;
	int* dev_out = NULL;

    hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	checkCudaErrors(cudaStatus);

	const int matrixSizeBytes = ROWS * COLUMNS * sizeof(int);

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_in, matrixSizeBytes);
	checkCudaErrors(cudaStatus);

	cudaStatus = hipMalloc((void**)&dev_out, matrixSizeBytes);
	checkCudaErrors(cudaStatus);

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_in, hostMatrix, matrixSizeBytes, hipMemcpyHostToDevice);
	checkCudaErrors(cudaStatus);

	//printMatrix((int*)hostMatrix, ROWS, COLUMNS);

	const int TRIALS = 1000;
	float elapsed;
	
	startTimer();
	for (int i = 0; i < TRIALS; i++) {
		// Launch a kernel on the GPU with one thread for each element.
		dim3 blockOrg;
		blockOrg.x = COLUMNS;
		blockOrg.y = ROWS;
		transpose <<<1, blockOrg >>> (dev_in, dev_out);
		cudaStatus = hipDeviceSynchronize();
		checkCudaErrors(cudaStatus);
	}
	elapsed = stopTimer();
	printf("%d transpose() trials took %f ms, %f ms/trial\r\n", TRIALS, elapsed, elapsed/TRIALS);
	

	printf("\r\n");
	startTimer();
	for (int i = 0; i < TRIALS; i++) {
		// Launch a kernel on the GPU with one thread for each element.
		dim3 blockOrg;
		blockOrg.x = COLUMNS;
		blockOrg.y = ROWS;
		transposeShared <<<1, blockOrg >>> (dev_in, dev_out);
		cudaStatus = hipDeviceSynchronize();
		checkCudaErrors(cudaStatus);
	}
	elapsed = stopTimer();
	printf("%d transposeShared() trials took %f ms, %f ms/trial\r\n", TRIALS, elapsed, elapsed / TRIALS);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	checkCudaErrors(cudaStatus);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	checkCudaErrors(cudaStatus);

	// Copy output vectors from GPU to host
	cudaStatus = hipMemcpy(hostMatrix, dev_out, matrixSizeBytes, hipMemcpyDeviceToHost);
	checkCudaErrors(cudaStatus);

	//printMatrix((int*)hostMatrix, COLUMNS, ROWS);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
	checkCudaErrors(cudaStatus);

	hipFree(dev_in);
	hipFree(dev_out);

    return 0;
}
