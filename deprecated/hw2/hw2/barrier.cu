#include "hip/hip_runtime.h"
#include "lock.cuh"
#include "barrier.cuh"
#include "timer.cuh"

#include <stdio.h>
#include <stdint.h>

#define NUM_BLOCKS 6
#define WARPS_PER_BLOCK 4 // max of 32
#define NUM_WARPS (NUM_BLOCKS * WARPS_PER_BLOCK)
const unsigned WARP_SIZE = 32;

/** the interface to a barrier */
class IBarrier {
protected:
	const unsigned m_expected;
	unsigned arrived;
	bool sense;

public:
	/** Initialize the barrier */
	__device__ IBarrier(const unsigned count) : m_expected(count) {
		arrived = 0;
		sense = true;
	}
	__device__ virtual void wait() = 0;
};

class SpinBarrier : public IBarrier, public WarpLevelLock {
public:
	__device__ SpinBarrier(const unsigned count) : IBarrier(count) {}

	__device__ virtual void wait() {
		// TODO: PART 5
		
	}
};

class TwoLevelBarrier : public SpinBarrier {
public:
	__device__ TwoLevelBarrier(const unsigned count) : SpinBarrier(count) {}

	__device__ virtual void wait() {
		// TODO: PART 6

	}
};

__device__ SpinBarrier* d_SpinBar = NULL;
__device__ TwoLevelBarrier* d_2LBar = NULL;

__global__ void initBarriers() {
	assert(blockIdx.x == 0 && threadIdx.x == 0);
	d_SpinBar = new SpinBarrier(NUM_WARPS);
	d_2LBar = new TwoLevelBarrier(NUM_BLOCKS);
}

__global__ void destroyBarriers() {
	assert(blockIdx.x == 0 && threadIdx.x == 0);
	delete d_SpinBar;
	delete d_2LBar;
}

/**
 * @param flavor what kind of global barrier to use
 * @param array pointer to 2D square array
 * @param arrayDim the width (and also height) of array
 * @param sourceRow for kernel launch barriers, which row to copy values from. Ignored for other barrier flavors
 */
__global__ void rotateRows(const BarrierFlavor flavor, int* array, const int arrayDim, const int sourceRow) {
	// only one thread per warp runs this code to avoid branch divergence issues
	//if (threadIdx.x % warpSize != 0) { return; }

	int in_c = ((blockDim.x * blockIdx.x) + threadIdx.x) /* warpSize*/;
	int out_c = (in_c + 1) % arrayDim;

	for (int r = 0; r < arrayDim - 1; r++) {
		// copy (row r, column c) to (row r+1, column c+1)
		if (flavor == KERNEL_LAUNCH_BARRIER) {
			r = sourceRow;
		}
		array[((r + 1) * arrayDim) + out_c] = array[(r * arrayDim) + in_c];
		if (flavor == KERNEL_LAUNCH_BARRIER) {
			return;
		} else if (flavor == SPIN_BARRIER) {
			d_SpinBar->wait();
		}
		else if (flavor == TWO_LEVEL_BARRIER) {
			d_2LBar->wait();
		} else {
			assert(false);
		}
	}
}

const char* barrierName(const BarrierFlavor flavor) {
	switch (flavor) {
	case KERNEL_LAUNCH_BARRIER: return "KernelLaunchBarrier";
	case SPIN_BARRIER: return "SpinBarrier";
	case TWO_LEVEL_BARRIER: return "2LevelBarrier";
	default: return "<invalid barrier>";
	}
}

void barrierTest(const BarrierFlavor flavor) {
	hipError_t cudaStatus;
	CudaTimer timer;

	const unsigned numThreads = NUM_BLOCKS * WARPS_PER_BLOCK * WARP_SIZE;
	const unsigned arraySizeBytes = numThreads * numThreads * sizeof(int);

	// ALLOCATE DEVICE MEMORY
	timer.start();

	int* h_array = new int[numThreads * numThreads];
	memset(h_array, 0, arraySizeBytes);

	uint64_t rowSum = 0;
	for (int i = 0; i < numThreads; i++) { // initialize first row with random values
		h_array[i] = rand();
		rowSum += h_array[i];
		//printf("%d ", h_array[i]);
	}
	//printf(" // initialized row with sum of %llu\n", rowSum);
	int* d_array;
	cudaStatus = hipMalloc(&d_array, arraySizeBytes);
	checkCudaErrors(cudaStatus);
	cudaStatus = hipMemcpy(d_array, h_array, arraySizeBytes, hipMemcpyHostToDevice);
	checkCudaErrors(cudaStatus);

	initBarriers << <1, 1 >> >();
	cudaStatus = hipGetLastError();
	checkCudaErrors(cudaStatus);

	printf("Setup device memory:  %3.1f ms \n", timer.stop());

	// LAUNCH KERNEL

	timer.start();
	if (flavor == KERNEL_LAUNCH_BARRIER) {
		// TODO: PART 4
		
	} else {
		rotateRows << <NUM_BLOCKS, WARPS_PER_BLOCK * WARP_SIZE >> >(flavor, d_array, numThreads, -1);
	}
	

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	checkCudaErrors(cudaStatus);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	checkCudaErrors(cudaStatus);

	printf("%s kernel time:  %3.1f ms \n", barrierName(flavor), timer.stop());

	// COPY ARRAY BACK TO HOST
	timer.start();
	cudaStatus = hipMemcpy(h_array, d_array, arraySizeBytes, hipMemcpyDeviceToHost);
	checkCudaErrors(cudaStatus);

	printf("Copy from device:  %3.1f ms \n", timer.stop());

	// CHECK ARRAY VALUES ARE CORRECT
	// each row and column should have the same sum
	const uint64_t expected = rowSum;
	bool foundError = false;

	// check rows
	for (int i = 0; i < numThreads; i++) {
		uint64_t sum = 0;
		for (int j = 0; j < numThreads; j++) {
			sum += h_array[(i * numThreads) + j];
		}
		if (sum != expected) {
			printf("Expected row %d to have sum of %llu BUT GOT %llu INSTEAD :-(\n", i, expected, sum);
			foundError = true;
		}
	}
	// check columns
	for (int i = 0; i < numThreads; i++) {
		uint64_t sum = 0;
		for (int j = 0; j < numThreads; j++) {
			sum += h_array[(j * numThreads) + i];
		}
		if (sum != expected) {
			printf("Expected column %d to have sum of %llu BUT GOT %llu INSTEAD :-(\n", i, expected, sum);
			foundError = true;
		}
	}

	/* print h_array for debugging
	for (int i = 0; i < numThreads; i++) {
	for (int j = 0; j < numThreads; j++) {
	printf("%d ", h_array[(i * numThreads) + j]);
	}
	printf("\n");
	}
	*/

	if (!foundError) {
		printf("Each row/column has expected sum of %llu\n", expected);
	}

	// CLEANUP

	destroyBarriers << <1, 1 >> >();
	cudaStatus = hipGetLastError();
	checkCudaErrors(cudaStatus);

	cudaStatus = hipFree(d_array);
	checkCudaErrors(cudaStatus);
	delete h_array;
}